#include "hip/hip_runtime.h"
#include "vec3.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <iostream>

__host__ __device__ vec3 reflect(const vec3& v, const vec3& n) {
	return v - 2 * dot(v, n)*n;
}

__host__ __device__ bool refract(const vec3& v, const vec3& n, float ni_over_nt, vec3& refracted) {
	vec3 uv = unit_vector(v);
	float dt = dot(uv, n);
	float discriminant = 1.0f - ni_over_nt * ni_over_nt*(1 - dt * dt);
	if (discriminant > 0) {
		refracted = ni_over_nt * (uv - n * dt) - n * sqrt(discriminant);
		return true;
	}
	else
		return false;
}

__host__ __device__ float schlick(float cosine, float ref_idx) {
	float r0 = (1 - ref_idx) / (1 + ref_idx);
	r0 = r0 * r0;
	return r0 + (1 - r0)*pow((1 - cosine), 5);
}
