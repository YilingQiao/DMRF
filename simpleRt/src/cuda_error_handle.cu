#include "cuda_error_handle.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

//Check the error code when recalling a CUDA function
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		std::cerr << hipGetErrorString((hipError_t)(static_cast<unsigned int>(result))) << std::endl;
		hipDeviceReset();
		exit(99);
	}
}